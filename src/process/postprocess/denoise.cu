#include "hip/hip_runtime.h"
#include "denoise.h"

DenoiseProcess::DenoiseProcess(int _w, int _h, shared_ptr<Scene> _s)
    : RenderProcess(_w, _h, _s), denoised(_w * _h), intensity(1)
{
    checkCudaErrors(hipStreamCreate(&stream));

    hipCtx_t cuda_context;
    hipError_t cu_res = hipCtxGetCurrent(&cuda_context);
    if (cu_res != hipSuccess)
    {
        cout << "ERROR::Failed to get current CUDA context" << endl;
        exit(-1);
    }

    OPTIX_CHECK(optixDeviceContextCreate(cuda_context, 0, &context));

    OptixDenoiserOptions options = {};
    OPTIX_CHECK(optixDenoiserCreate(context, OPTIX_DENOISER_MODEL_KIND_HDR, &options, &denoiser));

    OPTIX_CHECK(optixDenoiserComputeMemoryResources(denoiser, width, height, &denoiser_sizes));

    denoiser_state.resize(denoiser_sizes.stateSizeInBytes);
    denoiser_scratch.resize(denoiser_sizes.withoutOverlapScratchSizeInBytes);
    OPTIX_CHECK(optixDenoiserSetup(
        denoiser, 
        stream, 
        width, height,
        (hipDeviceptr_t)denoiser_state.data(),
        denoiser_sizes.stateSizeInBytes,
        (hipDeviceptr_t)denoiser_scratch.data(),
        denoiser_sizes.withoutOverlapScratchSizeInBytes
    ));
}

DenoiseProcess::~DenoiseProcess()
{
    OPTIX_CHECK(optixDenoiserDestroy(denoiser));
}

void DenoiseProcess::render(shared_ptr<Film> film)
{
    if(!enable) return;

    OptixImage2D input_layer;
    input_layer.data = (hipDeviceptr_t)film->get_pixels();
    input_layer.width = width;
    input_layer.height = height;
    input_layer.rowStrideInBytes = width * sizeof(float4);
    input_layer.pixelStrideInBytes = sizeof(float4);
    input_layer.format = OPTIX_PIXEL_FORMAT_FLOAT4;

    OptixImage2D output_layer;
    output_layer.data = (hipDeviceptr_t)denoised.data();
    output_layer.width = width;
    output_layer.height = height;
    output_layer.rowStrideInBytes = width * sizeof(float4);
    output_layer.pixelStrideInBytes = sizeof(float4);
    output_layer.format = OPTIX_PIXEL_FORMAT_FLOAT4;

    OPTIX_CHECK(optixDenoiserComputeIntensity(
        denoiser,
        stream,
        &input_layer,
        (hipDeviceptr_t)intensity.data(),
        (hipDeviceptr_t)denoiser_scratch.data(),
        denoiser_sizes.withoutOverlapScratchSizeInBytes
    ));

    OptixDenoiserParams params = {};
    params.denoiseAlpha = 1;
    params.hdrIntensity = (hipDeviceptr_t)intensity.data();
    params.blendFactor = 0;

    OptixDenoiserGuideLayer guide_layer = {};
    OptixDenoiserLayer layer = {};
    layer.input = input_layer;
    layer.output = output_layer;

    OPTIX_CHECK(optixDenoiserInvoke(
        denoiser,
        stream,
        &params,
        (hipDeviceptr_t)denoiser_state.data(),
        denoiser_sizes.stateSizeInBytes,
        &guide_layer,
        &layer,
        1, 0, 0,
        (hipDeviceptr_t)denoiser_scratch.data(),
        denoiser_sizes.withoutOverlapScratchSizeInBytes
    ));

    checkCudaErrors(hipMemcpy(film->get_pixels(), denoised.data(), width * height * sizeof(float4), hipMemcpyDeviceToDevice));
}

void DenoiseProcess::render_ui()
{
    ImGui::Checkbox("Denoise", &enable);
}