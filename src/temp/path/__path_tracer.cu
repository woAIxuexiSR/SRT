
#include <hip/hip_runtime.h>
// #include <optix_device.h>
// #include <device_launch_parameters.h>

// #include "launch_params/launch_params.h"
// #include "helper_optix.h"
// #include "my_math.h"

// extern "C" __constant__ LaunchParams<int> params;

// template<class T>
// static __forceinline__ __device__ T* getPRD()
// {
//     const unsigned u0 = optixGetPayload_0();
//     const unsigned u1 = optixGetPayload_1();
//     return reinterpret_cast<T*>(unpack_pointer(u0, u1));
// }

// extern "C" __global__ void __closesthit__radiance()
// {
//     const HitgroupData& sbtData = *(HitgroupData*)optixGetSbtDataPointer();
//     const int prim_idx = optixGetPrimitiveIndex();
//     const float2 uv = optixGetTriangleBarycentrics();
//     const float3 ray_dir = optixGetWorldRayDirection();

//     const uint3& index = sbtData.index[prim_idx];
//     const float3& v0 = sbtData.vertex[index.x];
//     const float3& v1 = sbtData.vertex[index.y];
//     const float3& v2 = sbtData.vertex[index.z];

//     HitInfo& prd = *(HitInfo*)getPRD<HitInfo>();
//     prd.hit = true;
//     prd.pos = v0 * (1.0f - uv.x - uv.y) + v1 * uv.x + v2 * uv.y;
//     prd.mat = &sbtData.mat;
//     prd.color = sbtData.mat.get_color();

//     float3 norm;
//     if (sbtData.normal)
//         norm = sbtData.normal[index.x] * (1.0f - uv.x - uv.y) + sbtData.normal[index.y] * uv.x + sbtData.normal[index.z] * uv.y;
//     else
//         norm = cross(v1 - v0, v2 - v0);
//     prd.normal = normalize(norm);
//     if (!(prd.mat->is_glass() || prd.mat->is_light()) && dot(prd.normal, ray_dir) > 0.0f)
//         prd.normal = -prd.normal;

//     if (sbtData.has_texture && sbtData.texcoord)
//     {
//         float2 tc = sbtData.texcoord[index.x] * (1.0f - uv.x - uv.y) + sbtData.texcoord[index.y] * uv.x + sbtData.texcoord[index.z] * uv.y;
//         float4 tex = tex2D<float4>(sbtData.texture, tc.x, tc.y);
//         prd.color = make_float3(tex.x, tex.y, tex.z);
//     }
// }

// extern "C" __global__ void __closesthit__shadow() {}

// extern "C" __global__ void __anyhit__radiance() {}

// extern "C" __global__ void __anyhit__shadow() {}

// extern "C" __global__ void __miss__radiance()
// {
//     HitInfo& prd = *(HitInfo*)getPRD<HitInfo>();
//     prd.hit = false;
// }

// extern "C" __global__ void __miss__shadow() {}

// extern "C" __global__ void __raygen__()
// {
//     const uint3 launch_idx = optixGetLaunchIndex();
//     const int ix = launch_idx.x, iy = launch_idx.y;

//     RandomGenerator rng(params.frame * params.height + iy, ix);
//     Camera& camera = params.camera;

//     HitInfo info;
//     thrust::pair<unsigned, unsigned> u = pack_pointer(&info);

//     float3 result = make_float3(0.0f);
//     for (int i = 0; i < params.spp; i++)
//     {
//         float xx = (ix + rng.random_float()) / params.width;
//         float yy = (iy + rng.random_float()) / params.height;
//         Ray ray = camera.get_ray(xx, yy);

//         float3 L = make_float3(0.0f), beta = make_float3(1.0f);
//         for (int depth = 0; depth < MAX_DEPTH; depth++)
//         {
//             optixTrace(params.traversable, ray.pos, ray.dir, 1e-3f, 1e16f, 0.0f,
//                 OptixVisibilityMask(255), OPTIX_RAY_FLAG_DISABLE_ANYHIT,
//                 RADIANCE_RAY_TYPE, RAY_TYPE_COUNT, RADIANCE_RAY_TYPE,
//                 u.first, u.second);

//             if (info.hit && info.mat->is_light())
//             {
//                 if (dot(info.normal, ray.dir) < 0.0f)
//                     L += beta * info.mat->emission();
//                 break;
//             }
//             if (!info.hit)
//             {
//                 L += beta * params.background;
//                 break;
//             }

//             MaterialSample ms = info.mat->sample(-ray.dir, info.normal, rng.random_float2(), info.color);
//             if (ms.pdf <= 1e-6f) break;
//             beta *= ms.f * abs(dot(ms.wi, info.normal)) / ms.pdf;
//             ray = Ray(info.pos, ms.wi);

//             if (depth >= 3)
//             {
//                 float p = max(beta.x, max(beta.y, beta.z));
//                 if (rng.random_float() > p) break;
//                 beta /= p;
//             }
//         }
//         result += L / params.spp;
//     }

//     int idx = iy * params.width + ix;
//     params.buffer[idx] = make_float4(result, 1.0f);
// }