#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <>

#include "helper_optix.h"
#include "my_params.h"
#include "my_math.h"

extern "C" __constant__ SimpleParams params;

template<class T>
__device__ inline T* getPRD()
{
    const unsigned u0 = optixGetPayload_0();
    const unsigned u1 = optixGetPayload_1();
    return reinterpret_cast<T*>(unpack_pointer(u0, u1));
}

extern "C" __global__ void __closesthit__radiance()
{
    const HitgroupData& sbtData = *(HitgroupData*)optixGetSbtDataPointer();
    const int prim_idx = optixGetPrimitiveIndex();
    const float2 uv = optixGetTriangleBarycentrics();
    const float3 ray_dir = optixGetWorldRayDirection();

    const Transform* transform = sbtData.instance->transform;
    const GTriangleMesh* mesh = sbtData.instance->mesh;

    const uint3& index = mesh->indices[prim_idx];
    const float3& v0 = mesh->vertices[index.x];
    const float3& v1 = mesh->vertices[index.y];
    const float3& v2 = mesh->vertices[index.z];

    HitInfo& prd = *(HitInfo*)getPRD<HitInfo>();
    prd.hit = true;
    prd.pos = transform->apply_point(v0 * (1.0f - uv.x - uv.y) + v1 * uv.x + v2 * uv.y);

    prd.normal = transform->apply_vector(cross(v1 - v0, v2 - v0));
    if (dot(prd.normal, ray_dir) > 0.0f) prd.normal = -prd.normal;

    float2 texcoord = uv;
    if (mesh->texcoords)
        texcoord = mesh->texcoords[index.x] * (1.0f - uv.x - uv.y) + mesh->texcoords[index.y] * uv.x + mesh->texcoords[index.z] * uv.y;
    float3 shading_normal = mesh->material->shading_normal(prd.normal, texcoord);
    shading_normal = transform->apply_vector(shading_normal);

    float3 tangent = shading_normal.x > 0.1f ? make_float3(0.0f, 1.0f, 0.0f) : make_float3(1.0f, 0.0f, 0.0f);
    if (mesh->tangents)
        tangent = transform->apply_vector(mesh->tangents[index.x] * (1.0f - uv.x - uv.y) + mesh->tangents[index.y] * uv.x + mesh->tangents[index.z] * uv.y);
    prd.onb = Onb(shading_normal, tangent);

    prd.color = mesh->material->surface_color(texcoord);
    prd.mat = mesh->material;

    prd.light_id = sbtData.light_id;
}

extern "C" __global__ void __closesthit__shadow() {}

extern "C" __global__ void __miss__radiance()
{
    HitInfo& prd = *(HitInfo*)getPRD<HitInfo>();
    prd.hit = false;
}

extern "C" __global__ void __miss__shadow() {}

extern "C" __global__ void __raygen__()
{
    const uint3 launch_idx = optixGetLaunchIndex();
    const int idx = launch_idx.x;

    const int ix = idx % params.width, iy = idx / params.width;

    RandomGenerator rng(params.seed + idx, 0);
    Camera& camera = params.camera;

    HitInfo info;
    uint2 u = pack_pointer(&info);

    float3 result = make_float3(0.0f);
    for (int i = 0; i < params.samples_per_pixel; i++)
    {
        float xx = (ix + rng.random_float()) / params.width;
        float yy = (iy + rng.random_float()) / params.height;
        Ray ray = camera.get_ray(xx, yy, rng);

        optixTrace(params.traversable, ray.pos, ray.dir, 1e-3f, 1e16f, 0.0f,
            OptixVisibilityMask(255), OPTIX_RAY_FLAG_DISABLE_ANYHIT,
            RADIANCE_RAY_TYPE, RAY_TYPE_COUNT, RADIANCE_RAY_TYPE,
            u.x, u.y);

        float3 L = make_float3(0.0f);
        if (!info.hit) continue;
        switch (params.type)
        {
        case SimpleParams::Type::Depth:
        {
            float d = length(info.pos - camera.controller.pos);
            float ratio = (d - params.min_depth) / (params.max_depth - params.min_depth);
            L = make_float3(ratio * 0.8f + 0.2f);
            break;
        }
        case SimpleParams::Type::Normal:
            L = (info.normal + 1.0f) * 0.5f;
            break;
        case SimpleParams::Type::BaseColor:
            L = info.color;
            break;
        case SimpleParams::Type::Ambient:
            L = info.color * (dot(info.normal, -ray.dir) * 0.5f + 0.5f);
            break;
        case SimpleParams::Type::FaceOrientation:
            // L = info.inner ? make_float3(1.0f, 0.85f, 0.0f) : make_float3(0.34f, 0.73f, 0.76f);
            L = make_float3(1.0f, 0.85f, 0.0f);
            break;
        }

        result += L / params.samples_per_pixel;
    }

    params.pixels[idx] = make_float4(result, 1.0f);
}