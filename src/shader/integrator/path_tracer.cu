#include "hip/hip_runtime.h"
#include <optix_device.h>
#include <>

#include "helper_optix.h"
#include "my_params.h"
#include "my_math.h"

extern "C" __constant__ PathTracerParams params;

template<class T>
__device__ inline T* getPRD()
{
    const unsigned u0 = optixGetPayload_0();
    const unsigned u1 = optixGetPayload_1();
    return reinterpret_cast<T*>(unpack_pointer(u0, u1));
}

extern "C" __global__ void __closesthit__radiance()
{
    const HitgroupData& sbtData = *(HitgroupData*)optixGetSbtDataPointer();
    const int prim_idx = optixGetPrimitiveIndex();
    const float2 uv = optixGetTriangleBarycentrics();
    const float3 ray_dir = optixGetWorldRayDirection();

    const GTriangleMesh* mesh = sbtData.mesh;
    const Transform* transform = sbtData.transform;
    const int light_id = sbtData.light_id;

    HitInfo& prd = *(HitInfo*)getPRD<HitInfo>();
    get_hitinfo(prd, mesh, transform, prim_idx, uv, ray_dir, light_id);
}

extern "C" __global__ void __closesthit__shadow()
{
    int& prd = *getPRD<int>();
    prd = 0;
}

extern "C" __global__ void __miss__radiance()
{
    HitInfo& prd = *(HitInfo*)getPRD<HitInfo>();
    prd.hit = false;
}

extern "C" __global__ void __miss__shadow()
{
    int& prd = *getPRD<int>();
    prd = 1;
}

extern "C" __global__ void __raygen__()
{
    const uint3 launch_idx = optixGetLaunchIndex();
    const int idx = launch_idx.x;
    const int ix = idx % params.width, iy = idx / params.width;

    RandomGenerator rng(params.seed + idx, 0);
    const Camera& camera = params.camera;
    const Light* light = params.light;

    HitInfo info; int visible;
    uint2 u = pack_pointer(&info), v = pack_pointer(&visible);

    float3 result = make_float3(0.0f);
    for (int i = 0; i < params.samples_per_pixel; i++)
    {
        float xx = (ix + rng.random_float()) / params.width;
        float yy = (iy + rng.random_float()) / params.height;
        Ray ray = camera.get_ray(xx, yy, rng);

        bool specular = true;
        float scatter_pdf = 1.0f;
        float3 L = make_float3(0.0f), beta = make_float3(1.0f);
        for (int depth = 0; depth < params.max_depth; depth++)
        {
            optixTrace(params.traversable, ray.pos, ray.dir, 1e-3f, 1e16f, 0.0f,
                OptixVisibilityMask(255), OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                RADIANCE_RAY_TYPE, RAY_TYPE_COUNT, RADIANCE_RAY_TYPE,
                u.x, u.y);

            // miss
            if (!info.hit)
            {
                L += beta * light->environment_emission(ray.dir);
                break;
            }

            // hit light
            if (info.hit && info.mat->is_emissive())
            {
                float cos_i = dot(info.normal, -ray.dir);
                if (cos_i <= 1e-4f) break;

                float mis_weight = 1.0f;
                if (params.use_nee && !specular)
                {
                    mis_weight = 0.0f;
                    if (params.use_mis)
                    {
                        float t2 = dot(info.pos - ray.pos, info.pos - ray.pos);
                        float light_pdf = light->sample_pdf(info.light_id) * t2 / cos_i;
                        mis_weight = scatter_pdf / (light_pdf + scatter_pdf);
                    }
                }
                L += beta * info.mat->emission(info.texcoord) * mis_weight;
                break;
            }

            // next event estimation
            if (params.use_nee && !info.mat->is_specular())
            {
                LightSample ls = light->sample(rng.random_float2());
                Ray shadow_ray(info.pos, normalize(ls.pos - info.pos));

                float cos_i = dot(info.normal, -ray.dir);
                if (cos_i < 0.0f) info.normal = -info.normal;

                float cos_o = dot(info.normal, shadow_ray.dir);
                float cos_light = dot(ls.normal, -shadow_ray.dir);
                float t = length(ls.pos - info.pos);
                if ((cos_light > 1e-4f) && (ls.pdf > 1e-4f) && (t > 1e-3f)
                    && (cos_o > 0.0f || info.mat->is_transmissive()))
                {
                    optixTrace(params.traversable, shadow_ray.pos, shadow_ray.dir, 1e-3f, t - 1e-3f, 0.0f,
                        OptixVisibilityMask(255), OPTIX_RAY_FLAG_DISABLE_ANYHIT,
                        SHADOW_RAY_TYPE, RAY_TYPE_COUNT, SHADOW_RAY_TYPE,
                        v.x, v.y);

                    if (visible)
                    {
                        float mis_weight = 1.0f;
                        float light_pdf = ls.pdf * t * t / cos_light;
                        if (params.use_mis)
                        {
                            float mat_pdf = info.mat->sample_pdf(shadow_ray.dir, -ray.dir, info.onb, info.color);
                            mis_weight = light_pdf / (light_pdf + mat_pdf);
                        }
                        L += beta * info.mat->eval(shadow_ray.dir, -ray.dir, info.onb, info.color)
                            * abs(cos_o) * ls.emission * mis_weight / light_pdf;
                    }
                }
            }

            // sample next direction
            BxDFSample ms = info.mat->sample(-ray.dir, rng.random_float2(), info.onb, info.color);
            if (ms.pdf <= 1e-4f) break;
            beta *= ms.f * ms.cos_theta / ms.pdf;

            specular = info.mat->is_specular();
            ray = Ray(info.pos, ms.wi);
            scatter_pdf = ms.pdf;

            // russian roulette
            if (depth >= params.rr_depth)
            {
                float p = max(max(beta.x, max(beta.y, beta.z)), 0.05f);
                if (rng.random_float() > p) break;
                beta /= p;
            }
        }

        if (check_valid(L.x) && check_valid(L.y) && check_valid(L.z))
            result += L / params.samples_per_pixel;
    }

    params.pixels[idx] = make_float4(result, 1.0f);
}