#include "hip/hip_runtime.h"
#include "scene.h"

/* build functions  */

int Scene::add_material(shared_ptr<Material> material)
{
    int id = materials.size();
    if (material->name == "")
        material->name = "material_" + std::to_string(id);
    materials.push_back(material);
    return id;
}

int Scene::find_material(const string& name)
{
    for (int i = 0; i < materials.size(); i++)
        if (materials[i]->name == name)
            return i;
    return -1;
}

int Scene::add_texture(shared_ptr<Texture> texture)
{
    int id = textures.size();
    if (texture->name == "")
        texture->name = "texture_" + std::to_string(id);
    textures.push_back(texture);
    return id;
}

int Scene::find_texture(const string& name)
{
    for (int i = 0; i < textures.size(); i++)
        if (textures[i]->name == name)
            return i;
    return -1;
}

int Scene::add_animation(shared_ptr<Animation> animation)
{
    int id = animations.size();
    animations.push_back(animation);
    return id;
}

int Scene::find_bone(const string& name)
{
    for (int i = 0; i < bones.size(); i++)
        if (bones[i].name == name)
            return i;
    return -1;
}

int Scene::add_bone(const Bone& bone)
{
    int id = bones.size();
    bones.push_back(bone);
    bone_transforms.push_back(Transform());
    return id;
}

/* GPU build functions */

void Scene::build_gscene()
{
    build_gscene_textures();
    build_gscene_materials();
    build_gscene_meshes();
    build_gscene_instances();
    build_gscene_lights();
}

void Scene::build_gscene_textures()
{
    int num_textures = (int)textures.size();

    gscene.texture_arrays.resize(num_textures);
    gscene.texture_objects.resize(num_textures);

    for (int i = 0; i < num_textures; i++)
    {
        auto& image = textures[i]->image;
        hipChannelFormatDesc channel_desc = hipCreateChannelDesc<uchar4>();
        int width = image.resolution.x;
        int height = image.resolution.y;
        int pitch = width * sizeof(uchar4);
        if (image.format == Image::Format::Float)
        {
            channel_desc = hipCreateChannelDesc<float4>();
            pitch = width * sizeof(float4);
        }

        hipArray_t& pixel_array = gscene.texture_arrays[i];
        checkCudaErrors(hipMallocArray(&pixel_array, &channel_desc, width, height));
        checkCudaErrors(hipMemcpy2DToArray(pixel_array, 0, 0, image.get_pixels(), pitch, pitch, height, hipMemcpyHostToDevice));

        hipResourceDesc res_desc = {};
        res_desc.resType = hipResourceTypeArray;
        res_desc.res.array.array = pixel_array;

        hipTextureDesc tex_desc = {};
        tex_desc.addressMode[0] = hipAddressModeWrap;
        tex_desc.addressMode[1] = hipAddressModeWrap;
        tex_desc.filterMode = hipFilterModeLinear;
        tex_desc.readMode = hipReadModeNormalizedFloat;
        if (image.format == Image::Format::Float)
            tex_desc.readMode = hipReadModeElementType;
        tex_desc.normalizedCoords = 1;
        tex_desc.maxAnisotropy = 1;
        tex_desc.maxMipmapLevelClamp = 99;
        tex_desc.minMipmapLevelClamp = 0;
        tex_desc.mipmapFilterMode = hipFilterModePoint;
        tex_desc.borderColor[0] = 1.0f;
        tex_desc.sRGB = 1;
        if (image.format == Image::Format::Float)
            tex_desc.sRGB = 0;

        checkCudaErrors(hipCreateTextureObject(&gscene.texture_objects[i], &res_desc, &tex_desc, nullptr));
    }
}

void Scene::build_gscene_materials()
{
    int num_materials = (int)materials.size();

    vector<GMaterial> gmaterials(num_materials);
    for (int i = 0; i < num_materials; i++)
    {
        gmaterials[i].bxdf = materials[i]->bxdf;
        gmaterials[i].base_color = materials[i]->base_color;
        gmaterials[i].emission_color = materials[i]->emission_color;
        gmaterials[i].intensity = materials[i]->intensity;
        if (materials[i]->color_tex_id != -1)
            gmaterials[i].color_tex = gscene.texture_objects[materials[i]->color_tex_id];
        if (materials[i]->normal_tex_id != -1)
            gmaterials[i].normal_tex = gscene.texture_objects[materials[i]->normal_tex_id];
    }
    gscene.material_buffer.resize_and_copy_from_host(gmaterials);
}

void Scene::build_gscene_meshes()
{
    int mesh_num = (int)meshes.size();

    gscene.vertex_buffer.resize(mesh_num);
    gscene.index_buffer.resize(mesh_num);
    gscene.normal_buffer.resize(mesh_num);
    gscene.tangent_buffer.resize(mesh_num);
    gscene.texcoord_buffer.resize(mesh_num);

#ifndef SRT_HIGH_PERFORMANCE
    gscene.original_vertex_buffer.resize(mesh_num);
    gscene.original_normal_buffer.resize(mesh_num);
    gscene.original_tangent_buffer.resize(mesh_num);
    gscene.bone_id_buffer.resize(mesh_num);
    gscene.bone_weight_buffer.resize(mesh_num);
    if (!bones.empty())
        gscene.bone_transform_buffer.resize_and_copy_from_host(bone_transforms);
#endif

    for (int i = 0; i < mesh_num; i++)
    {
        auto mesh = meshes[i];

        gscene.vertex_buffer[i].resize_and_copy_from_host(mesh->vertices);
        gscene.index_buffer[i].resize_and_copy_from_host(mesh->indices);
        if (!meshes[i]->normals.empty())
            gscene.normal_buffer[i].resize_and_copy_from_host(mesh->normals);
        if (!meshes[i]->tangents.empty())
            gscene.tangent_buffer[i].resize_and_copy_from_host(mesh->tangents);
        if (!meshes[i]->texcoords.empty())
            gscene.texcoord_buffer[i].resize_and_copy_from_host(mesh->texcoords);

#ifndef SRT_HIGH_PERFORMANCE
        if (mesh->has_bone)
        {
            gscene.original_vertex_buffer[i].resize_and_copy_from_host(mesh->vertices);
            if (!meshes[i]->normals.empty())
                gscene.original_normal_buffer[i].resize_and_copy_from_host(mesh->normals);
            if (!meshes[i]->tangents.empty())
                gscene.original_tangent_buffer[i].resize_and_copy_from_host(mesh->tangents);
            gscene.bone_id_buffer[i].resize_and_copy_from_host(mesh->bone_ids);
            gscene.bone_weight_buffer[i].resize_and_copy_from_host(mesh->bone_weights);
        }
#endif
    }

    vector<GTriangleMesh> gmeshes(mesh_num);
    for (int i = 0; i < mesh_num; i++)
    {
        gmeshes[i].vertices = gscene.vertex_buffer[i].data();
        gmeshes[i].indices = gscene.index_buffer[i].data();
        gmeshes[i].normals = gscene.normal_buffer[i].data();
        gmeshes[i].tangents = gscene.tangent_buffer[i].data();
        gmeshes[i].texcoords = gscene.texcoord_buffer[i].data();

        gmeshes[i].material = gscene.material_buffer.data() + meshes[i]->material_id;
    }
    gscene.mesh_buffer.resize_and_copy_from_host(gmeshes);
}

void Scene::build_gscene_instances()
{
    gscene.instance_transform_buffer.resize_and_copy_from_host(instance_transforms);
}

void Scene::build_gscene_lights()
{
    gscene.instance_light_id.resize(instances.size(), -1);

    int num_light = 0;
    for (int i = 0; i < (int)instances.size(); i++)
    {
        auto material = materials[meshes[instances[i]]->material_id];
        if (material->intensity <= 0.0f) continue;

        gscene.instance_light_id[i] = num_light;
        num_light++;
    }

    gscene.light_area_buffer.resize(num_light);
    vector<AreaLight> area_lights(num_light);
    float weight_sum = 0.0f;
    for (int i = 0; i < (int)instances.size(); i++)
    {
        auto material = materials[meshes[instances[i]]->material_id];
        if (gscene.instance_light_id[i] == -1) continue;

        int light_id = gscene.instance_light_id[i];
        auto mesh = meshes[instances[i]];

        int face_num = (int)mesh->indices.size();
        float area = 0.0f;
        vector<float> areas(face_num);
        for (int j = 0; j < face_num; j++)
        {
            auto& index = mesh->indices[j];
            auto v0 = mesh->vertices[index.x], v1 = mesh->vertices[index.y], v2 = mesh->vertices[index.z];
            areas[j] = length(cross(v1 - v0, v2 - v0)) * 0.5f;
            area += areas[j];
        }
        weight_sum += area * material->intensity;
        gscene.light_area_buffer[light_id].resize_and_copy_from_host(areas);

        area_lights[light_id].mesh = gscene.mesh_buffer.data() + instances[i];
        area_lights[light_id].transform = gscene.instance_transform_buffer.data() + i;

        area_lights[light_id].face_num = face_num;
        area_lights[light_id].areas = gscene.light_area_buffer[light_id].data();
        area_lights[light_id].area_sum = area;
    }
    gscene.area_light_buffer.resize_and_copy_from_host(area_lights);

    EnvironmentLight env_light;
    if (environment_map_id == -1)
    {
        env_light.type = EnvironmentLight::Type::Constant;
        env_light.emission_color = background;
    }
    else
    {
        env_light.type = EnvironmentLight::Type::UVMap;
        env_light.texture = gscene.texture_objects[environment_map_id];
    }
    gscene.environment_light_buffer.resize_and_copy_from_host(&env_light, 1);

    Light light;
    light.num = num_light;
    light.lights = gscene.area_light_buffer.data();
    light.weight_sum = weight_sum;
    light.env_light = gscene.environment_light_buffer.data();
    gscene.light_buffer.resize_and_copy_from_host(&light, 1);
}


/* useful functions */

void Scene::compute_aabb()
{
    aabb = AABB();
    for (int i = 0; i < instances.size(); i++)
    {
        auto mesh = meshes[instances[i]];
        aabb.expand(instance_transforms[i].apply_aabb(mesh->aabb));
    }
}

void Scene::update(float t)
{
    if (!dynamic) return;
    update_node(root, t, Transform());
    update_gscene();
}

void Scene::update_node(shared_ptr<SceneGraphNode> node, float t, const Transform& parent_transform)
{
    Transform node_transform = node->transform;
    if (node->animation_id != -1)
    {
        // for (int i = 0; i < 4; i++)
        // {
        //     for (int j = 0; j < 4; j++)
        //         cout << node_transform[i][j] << " ";
        //     cout << endl;
        // }
        node_transform = animations[node->animation_id]->get_transform(t);
        // for (int i = 0; i < 4; i++)
        // {
        //     for (int j = 0; j < 4; j++)
        //         cout << node_transform[i][j] << " ";
        //     cout << endl;
        // }
        // exit(-1);
    }
    Transform global_transform = parent_transform * node_transform;

    for (auto id : node->instance_ids)
        instance_transforms[id] = global_transform;

#ifndef SRT_HIGH_PERFORMANCE
    if (node->bone_id != -1)
        bone_transforms[node->bone_id] = global_transform * bones[node->bone_id].offset;
#endif

    for (auto& child : node->children)
        update_node(child, t, global_transform);
}

void Scene::update_gscene()
{
    gscene.instance_transform_buffer.copy_from_host(instance_transforms);

#ifndef SRT_HIGH_PERFORMANCE
    if (bones.empty())
        return;

    gscene.bone_transform_buffer.copy_from_host(bone_transforms);
    for (int i = 0; i < (int)meshes.size(); i++)
    {
        if (!meshes[i]->has_bone)
            continue;

        int num_vertices = meshes[i]->vertices.size();
        int num_bones = bones.size();
        Transform* bone_transforms = gscene.bone_transform_buffer.data();
        int* bone_ids = gscene.bone_id_buffer[i].data();
        float* bone_weights = gscene.bone_weight_buffer[i].data();

        float3* vertices = gscene.vertex_buffer[i].data();
        float3* normals = gscene.normal_buffer[i].data();
        float3* tangents = gscene.tangent_buffer[i].data();
        float3* original_vertices = gscene.original_vertex_buffer[i].data();
        float3* original_normals = gscene.original_normal_buffer[i].data();
        float3* original_tangents = gscene.original_tangent_buffer[i].data();

        tcnn::parallel_for_gpu(num_vertices, [=] __device__(int idx) {

            vertices[idx] = make_float3(0.0f);
            if (normals) normals[idx] = make_float3(0.0f);
            if (tangents) tangents[idx] = make_float3(0.0f);

            for (int j = 0; j < MAX_BONE_PER_VERTEX; j++)
            {
                int bone_id = bone_ids[idx * MAX_BONE_PER_VERTEX + j];
                float bone_weight = bone_weights[idx * MAX_BONE_PER_VERTEX + j];

                if (bone_id == -1)
                {
                    if (j == 0)  // no bone assigned
                    {
                        vertices[idx] = original_vertices[idx];
                        if (normals) normals[idx] = original_normals[idx];
                        if (tangents) tangents[idx] = original_tangents[idx];
                    }
                    break;
                }
                if (bone_id >= num_bones)
                    continue;

                Transform& t = bone_transforms[bone_id];
                vertices[idx] += t.apply_point(original_vertices[idx]) * bone_weight;
                if (normals) normals[idx] += t.apply_vector(original_normals[idx]) * bone_weight;
                if (tangents) tangents[idx] += t.apply_vector(original_tangents[idx]) * bone_weight;
            }

        });
    }
    checkCudaErrors(hipDeviceSynchronize());
#endif
}

void Scene::render_ui()
{
    ImGui::Checkbox("Dynamic", &dynamic);
}