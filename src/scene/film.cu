#include "hip/hip_runtime.h"
#include "film.h"

__global__ void fToUchar_k(int n_elements, float4* __restrict__ src, uchar4* __restrict__ dst)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_elements) return;
    float4 clamped = clamp(src[i], 0.0f, 1.0f);
    float w = 1.0f / 2.2f;
    float4 hdr = make_float4(powf(clamped.x, w), powf(clamped.y, w), powf(clamped.z, w), powf(clamped.w, w));
    float4 v = hdr * 255.f;
    dst[i] = make_uchar4(v.x, v.y, v.z, v.w);
}

__global__ void flipfVertical_k(int n_elements, int width, int height, float4* __restrict__ src, float4* __restrict__ dst)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_elements) return;
    int x = i % width, y = i / width;
    dst[i] = src[(height - y - 1) * width + x];
}

void Film::fToUchar()
{
    int pixel_num = width * height;
    tcnn::linear_kernel(fToUchar_k, 0, 0, pixel_num, pixels_f.data(), pixels_u.data());
    checkCudaErrors(hipDeviceSynchronize());
}

void Film::save_jpg(const std::string& filename) const
{
    std::vector<uchar4> pixels_cpu(width * height);
    pixels_u.copy_to_host(pixels_cpu);

    stbi_flip_vertically_on_write(true);
    int ret = stbi_write_jpg(filename.c_str(), width, height, 4, (void*)pixels_cpu.data(), 100);
    if (ret == 0)
        std::cout << "Failed to save image: " << filename << std::endl;
}

void Film::save_exr(const std::string& filename) const
{
    EXRHeader header;
    InitEXRHeader(&header);
    EXRImage image;
    InitEXRImage(&image);

    GPUMemory<float4> flipped(width * height);
    tcnn::linear_kernel(flipfVertical_k, 0, 0, width * height, width, height, pixels_f.data(), flipped.data());
    std::vector<float4> pixels_cpu(width * height);
    flipped.copy_to_host(pixels_cpu);

    std::vector<float> images[4]{
        std::vector<float>(width * height),
        std::vector<float>(width * height),
        std::vector<float>(width * height),
        std::vector<float>(width * height)
    };
    for (int i = 0; i < width * height; i++)
    {
        images[0][i] = pixels_cpu[i].x;
        images[1][i] = pixels_cpu[i].y;
        images[2][i] = pixels_cpu[i].z;
        images[3][i] = pixels_cpu[i].w;
    }

    float* image_ptr[4];
    image_ptr[0] = &(images[2][0]);
    image_ptr[1] = &(images[1][0]);
    image_ptr[2] = &(images[0][0]);
    image_ptr[3] = &(images[3][0]);

    image.num_channels = 4;
    image.images = (unsigned char**)image_ptr;
    image.width = width;
    image.height = height;
    header.num_channels = 4;
    header.channels = (EXRChannelInfo*)malloc(sizeof(EXRChannelInfo) * header.num_channels);
    strncpy(header.channels[0].name, "B", 255); header.channels[0].name[strlen("B")] = '\0';
    strncpy(header.channels[1].name, "G", 255); header.channels[1].name[strlen("G")] = '\0';
    strncpy(header.channels[2].name, "R", 255); header.channels[2].name[strlen("R")] = '\0';
    strncpy(header.channels[3].name, "A", 255); header.channels[3].name[strlen("A")] = '\0';
    header.pixel_types = (int*)malloc(sizeof(int) * header.num_channels);
    header.requested_pixel_types = (int*)malloc(sizeof(int) * header.num_channels);
    for (int i = 0; i < header.num_channels; i++)
    {
        header.pixel_types[i] = TINYEXR_PIXELTYPE_FLOAT;
        header.requested_pixel_types[i] = TINYEXR_PIXELTYPE_HALF;
    }

    const char* err;
    int ret = SaveEXRImageToFile(&image, &header, filename.c_str(), &err);
    if (ret != TINYEXR_SUCCESS)
        std::cout << "Failed to save image: " << filename << std::endl;

    free(header.channels);
    free(header.pixel_types);
    free(header.requested_pixel_types);
}