#include "hip/hip_runtime.h"
#include "optixRayTracer.h"
#include <optix_function_table_definition.h>

std::string getPtxFromFile(const std::string& filename)
{
    std::ifstream fs(filename, std::ios::in);
    if (!fs.is_open()) {
        std::cerr << "Could not open file " << filename << std::endl;
        exit(1);
    }

    std::stringstream ss;
    ss << fs.rdbuf();
    std::string str = ss.str();
    fs.close();

    return str;
}

/* the only allowed shader function names */
const std::string raygenName = "__raygen__";
const std::string missName[2] = { "__miss__radiance", "__miss__shadow" };
const std::pair<std::string, std::string> hitgroupName[2] = {
    {"__closesthit__radiance", "__anyhit__radiance"},
    {"__closesthit__shadow", "__anyhit__shadow"}
};

void OptixRayTracer::initOptix()
{
    checkCudaErrors(hipFree(0));

    int numDevices;
    checkCudaErrors(hipGetDeviceCount(&numDevices));
    if (numDevices == 0)
    {
        std::cout << "no CUDA capable devices found!" << std::endl;
        exit(-1);
    }
    std::cout << "found " << numDevices << " CUDA devices" << std::endl;

    OPTIX_CHECK(optixInit());
    std::cout << "successfully initialized optix ..." << std::endl;
}

void OptixRayTracer::createContext()
{
    const int deviceID = 0;
    checkCudaErrors(hipSetDevice(deviceID));
    checkCudaErrors(hipStreamCreate(&stream));

    hipDeviceProp_t deviceProps;
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, deviceID));
    std::cout << "running on device " << deviceProps.name << std::endl;

    hipCtx_t cudaContext;
    hipError_t cuRes = hipCtxGetCurrent(&cudaContext);
    if (cuRes != hipSuccess)
    {
        std::cout << "could not get CUDA context" << std::endl;
        exit(-1);
    }

    OPTIX_CHECK(optixDeviceContextCreate(cudaContext, 0, &optixContext));
}

void OptixRayTracer::createModule(const std::string& ptx, OptixPipelineCompileOptions& pipelineCompileOptions, std::vector<OptixProgramGroup>& programGroups)
{
    // create module
    OptixModule module;

    OptixModuleCompileOptions moduleCompileOptions = {};
    moduleCompileOptions.maxRegisterCount = 50;
    moduleCompileOptions.optLevel = OPTIX_COMPILE_OPTIMIZATION_DEFAULT;
    moduleCompileOptions.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_NONE;
    // moduleCompileOptions.optLevel = OPTIX_COMPILE_OPTIMIZATION_LEVEL_0;
    // moduleCompileOptions.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_FULL;

    {
        char log[2048];
        size_t logSize = sizeof(log);
        OPTIX_CHECK(optixModuleCreateFromPTX(optixContext, &moduleCompileOptions, &pipelineCompileOptions, ptx.c_str(), ptx.size(), log, &logSize, &module));
        if (logSize > 1)
            std::cout << "Optix log : " << log << std::endl;
    }

    ModuleProgramGroup modulePG;

    // create raygen programs
    {
        OptixProgramGroupOptions pgOptions = {};
        OptixProgramGroupDesc pgDesc = {};

        pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
        pgDesc.raygen.module = module;
        pgDesc.raygen.entryFunctionName = raygenName.c_str();

        char log[2048];
        size_t logSize = sizeof(log);
        OPTIX_CHECK(optixProgramGroupCreate(optixContext, &pgDesc, 1, &pgOptions, log, &logSize, &modulePG.raygenPG));
        if (logSize > 1)
            std::cout << "Optix log : " << log << std::endl;

        programGroups.push_back(modulePG.raygenPG);
    }

    // create miss programs
    {
        for (int i = 0; i < 2; i++)
        {
            OptixProgramGroupOptions pgOptions = {};
            OptixProgramGroupDesc pgDesc = {};

            pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_MISS;
            pgDesc.miss.module = module;
            pgDesc.miss.entryFunctionName = missName[i].c_str();

            char log[2048];
            size_t logSize = sizeof(log);
            OPTIX_CHECK(optixProgramGroupCreate(optixContext, &pgDesc, 1, &pgOptions, log, &logSize, &modulePG.missPGs[i]));
            if (logSize > 1)
                std::cout << "Optix log : " << log << std::endl;

            programGroups.push_back(modulePG.missPGs[i]);
        }
    }

    // create hitgroup programs
    {
        for (int i = 0; i < 2; i++)
        {
            OptixProgramGroupOptions pgOptions = {};
            OptixProgramGroupDesc pgDesc = {};

            pgDesc.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
            pgDesc.hitgroup.moduleCH = module;
            pgDesc.hitgroup.entryFunctionNameCH = hitgroupName[i].first.c_str();
            pgDesc.hitgroup.moduleAH = module;
            pgDesc.hitgroup.entryFunctionNameAH = hitgroupName[i].second.c_str();

            char log[2048];
            size_t logSize = sizeof(log);
            OPTIX_CHECK(optixProgramGroupCreate(optixContext, &pgDesc, 1, &pgOptions, log, &logSize, &modulePG.hitgroupPGs[i]));
            if (logSize > 1)
                std::cout << "Optix log : " << log << std::endl;

            programGroups.push_back(modulePG.hitgroupPGs[i]);
        }
    }

    modulePGs.push_back(modulePG);
}

void OptixRayTracer::createPipelines()
{
    OptixPipelineCompileOptions pipelineCompileOptions = {};
    pipelineCompileOptions.traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;
    pipelineCompileOptions.usesMotionBlur = false;
    pipelineCompileOptions.numPayloadValues = 2;
    pipelineCompileOptions.numAttributeValues = 2;
    pipelineCompileOptions.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;
    pipelineCompileOptions.pipelineLaunchParamsVariableName = "launchParams";

    OptixPipelineLinkOptions pipelineLinkOptions = {};
    pipelineLinkOptions.maxTraceDepth = 2;

    pipelines.resize(modulePTXs.size());
    for (size_t i = 0; i < modulePTXs.size(); i++)
    {
        std::vector<OptixProgramGroup> programGroups;
        createModule(modulePTXs[i], pipelineCompileOptions, programGroups);

        char log[2048];
        size_t logSize = sizeof(log);
        OPTIX_CHECK(optixPipelineCreate(optixContext, &pipelineCompileOptions, &pipelineLinkOptions, programGroups.data(), (unsigned)programGroups.size(), log, &logSize, &pipelines[i]));
        if (logSize > 1)
            std::cout << "Optix log : " << log << std::endl;

        OPTIX_CHECK(optixPipelineSetStackSize(pipelines[i], 2048, 2048, 2048, 1));
    }
}

void OptixRayTracer::buildAccel()
{
    int meshNum = (int)model->meshes.size();

    vertexBuffer.resize(meshNum);
    indexBuffer.resize(meshNum);
    texcoordBuffer.resize(meshNum);
    normalBuffer.resize(meshNum);

    std::vector<OptixBuildInput> triangleInput(meshNum);
    std::vector<hipDeviceptr_t> d_vertices(meshNum);
    std::vector<hipDeviceptr_t> d_indices(meshNum);
    std::vector<uint32_t> triangleInputFlags(meshNum);

    for (int i = 0; i < meshNum; i++)
    {
        TriangleMesh& mesh = *(model->meshes[i]);
        vertexBuffer[i].resize_and_copy_from_host(mesh.vertices);
        indexBuffer[i].resize_and_copy_from_host(mesh.indices);
        if (!mesh.texcoords.empty())
            texcoordBuffer[i].resize_and_copy_from_host(mesh.texcoords);
        if (!mesh.normals.empty())
            normalBuffer[i].resize_and_copy_from_host(mesh.normals);

        triangleInput[i] = {};
        triangleInput[i].type = OPTIX_BUILD_INPUT_TYPE_TRIANGLES;

        d_vertices[i] = (hipDeviceptr_t)vertexBuffer[i].data();
        d_indices[i] = (hipDeviceptr_t)indexBuffer[i].data();

        triangleInput[i].triangleArray.vertexFormat = OPTIX_VERTEX_FORMAT_FLOAT3;
        triangleInput[i].triangleArray.vertexStrideInBytes = sizeof(float3);
        triangleInput[i].triangleArray.numVertices = (unsigned)mesh.vertices.size();
        triangleInput[i].triangleArray.vertexBuffers = &d_vertices[i];

        triangleInput[i].triangleArray.indexFormat = OPTIX_INDICES_FORMAT_UNSIGNED_INT3;
        triangleInput[i].triangleArray.indexStrideInBytes = sizeof(uint3);
        triangleInput[i].triangleArray.numIndexTriplets = (unsigned)mesh.indices.size();
        triangleInput[i].triangleArray.indexBuffer = d_indices[i];

        triangleInputFlags[i] = 0;
        triangleInput[i].triangleArray.flags = &triangleInputFlags[i];
        triangleInput[i].triangleArray.numSbtRecords = 1;
        triangleInput[i].triangleArray.sbtIndexOffsetBuffer = 0;
        triangleInput[i].triangleArray.sbtIndexOffsetSizeInBytes = 0;
        triangleInput[i].triangleArray.sbtIndexOffsetStrideInBytes = 0;
    }

    OptixAccelBuildOptions accelOptions = {};
    accelOptions.buildFlags = OPTIX_BUILD_FLAG_NONE | OPTIX_BUILD_FLAG_ALLOW_COMPACTION;
    accelOptions.motionOptions.numKeys = 1;
    accelOptions.operation = OPTIX_BUILD_OPERATION_BUILD;

    OptixAccelBufferSizes blasBufferSizes;
    OPTIX_CHECK(optixAccelComputeMemoryUsage(optixContext, &accelOptions, triangleInput.data(), meshNum, &blasBufferSizes));

    GPUMemory<uint64_t> compactedSizeBuffer(1);
    OptixAccelEmitDesc emitDesc;
    emitDesc.type = OPTIX_PROPERTY_TYPE_COMPACTED_SIZE;
    emitDesc.result = (hipDeviceptr_t)compactedSizeBuffer.data();

    GPUMemory<unsigned char> tempBuffer(blasBufferSizes.tempSizeInBytes);
    GPUMemory<unsigned char> outputBuffer(blasBufferSizes.outputSizeInBytes);

    OPTIX_CHECK(optixAccelBuild(
        optixContext,
        stream,
        &accelOptions,
        triangleInput.data(),
        meshNum,
        (hipDeviceptr_t)tempBuffer.data(),
        blasBufferSizes.tempSizeInBytes,
        (hipDeviceptr_t)outputBuffer.data(),
        blasBufferSizes.outputSizeInBytes,
        &traversable,
        &emitDesc,
        1));
    checkCudaErrors(hipDeviceSynchronize());

    uint64_t compactedSize;
    compactedSizeBuffer.copy_to_host(&compactedSize);

    asBuffer.resize(compactedSize);
    OPTIX_CHECK(optixAccelCompact(optixContext, stream, traversable, (hipDeviceptr_t)asBuffer.data(), compactedSize, &traversable));
    checkCudaErrors(hipDeviceSynchronize());
}

void OptixRayTracer::createTextures()
{
    int numTextures = (int)model->textures.size();

    textureArrays.resize(numTextures);
    textureObjects.resize(numTextures);

    for (int i = 0; i < numTextures; i++)
    {
        Texture* texture = model->textures[i];

        hipChannelFormatDesc channel_desc = hipCreateChannelDesc<uchar4>();

        int width = texture->resolution.x;
        int height = texture->resolution.y;
        int nComponents = 4;
        int pitch = width * nComponents * sizeof(unsigned char);

        hipArray_t& pixelArray = textureArrays[i];
        checkCudaErrors(hipMallocArray(&pixelArray, &channel_desc, width, height));
        checkCudaErrors(hipMemcpy2DToArray(pixelArray, 0, 0, texture->pixels, pitch, pitch, height, hipMemcpyHostToDevice));

        hipResourceDesc res_desc = {};
        res_desc.resType = hipResourceTypeArray;
        res_desc.res.array.array = pixelArray;

        hipTextureDesc tex_desc = {};
        tex_desc.addressMode[0] = hipAddressModeWrap;
        tex_desc.addressMode[1] = hipAddressModeWrap;
        tex_desc.filterMode = hipFilterModeLinear;
        tex_desc.readMode = hipReadModeNormalizedFloat;
        tex_desc.normalizedCoords = 1;
        tex_desc.maxAnisotropy = 1;
        tex_desc.maxMipmapLevelClamp = 99;
        tex_desc.minMipmapLevelClamp = 0;
        tex_desc.mipmapFilterMode = hipFilterModePoint;
        tex_desc.borderColor[0] = 1.0f;
        tex_desc.sRGB = 0;

        hipTextureObject_t cuda_tex = 0;
        checkCudaErrors(hipCreateTextureObject(&cuda_tex, &res_desc, &tex_desc, nullptr));
        textureObjects[i] = cuda_tex;
    }
}

void OptixRayTracer::buildSBT()
{
    sbts.resize(modulePGs.size());
    raygenRecordsBuffer.resize(modulePGs.size());
    missRecordsBuffer.resize(modulePGs.size());
    hitgroupRecordsBuffer.resize(modulePGs.size());

    for (int i = 0; i < modulePGs.size(); i++)
    {
        RaygenSBTRecord raygenRec;
        OPTIX_CHECK(optixSbtRecordPackHeader(modulePGs[i].raygenPG, &raygenRec));
        raygenRecordsBuffer[i].resize_and_copy_from_host(&raygenRec, 1);
        sbts[i].raygenRecord = (hipDeviceptr_t)raygenRecordsBuffer[i].data();

        std::vector<MissSBTRecord> missRecs;
        for (int j = 0; j < 2; j++)
        {
            MissSBTRecord rec;
            OPTIX_CHECK(optixSbtRecordPackHeader(modulePGs[i].missPGs[j], &rec));
            missRecs.push_back(rec);
        }
        missRecordsBuffer[i].resize_and_copy_from_host(missRecs);
        sbts[i].missRecordBase = (hipDeviceptr_t)missRecordsBuffer[i].data();
        sbts[i].missRecordStrideInBytes = sizeof(MissSBTRecord);
        sbts[i].missRecordCount = (unsigned)missRecs.size();

        int meshNum = (int)model->meshes.size();
        std::vector<HitgroupSBTRecord> hitgroupRecs;
        for (int k = 0; k < meshNum; k++)
        {
            for (int j = 0; j < 2; j++)
            {
                HitgroupSBTRecord rec;
                OPTIX_CHECK(optixSbtRecordPackHeader(modulePGs[i].hitgroupPGs[j], &rec));

                rec.data.vertex = (float3*)vertexBuffer[k].data();
                rec.data.index = (uint3*)indexBuffer[k].data();
                rec.data.normal = (float3*)normalBuffer[k].data();
                rec.data.texcoord = (float2*)texcoordBuffer[k].data();
                rec.data.mat = model->meshes[k]->mat;
                if (model->meshes[k]->textureId >= 0)
                {
                    rec.data.hasTexture = true;
                    rec.data.texture = textureObjects[model->meshes[k]->textureId];
                }
                else
                    rec.data.hasTexture = false;

                hitgroupRecs.push_back(rec);
            }
        }
        hitgroupRecordsBuffer[i].resize_and_copy_from_host(hitgroupRecs);
        sbts[i].hitgroupRecordBase = (hipDeviceptr_t)hitgroupRecordsBuffer[i].data();
        sbts[i].hitgroupRecordStrideInBytes = sizeof(HitgroupSBTRecord);
        sbts[i].hitgroupRecordCount = (unsigned)hitgroupRecs.size();
    }
}

void OptixRayTracer::generateLight()
{
    std::vector<float3> lightVertices;
    std::vector<float3> lightNormals;
    std::vector<uint3> lightIndices;
    std::vector<float3> lightEmissions;
    for (int i = 0; i < model->meshes.size(); i++)
    {
        TriangleMesh* mesh = model->meshes[i];
        if (mesh->mat.getType() != MaterialType::Emissive)
            continue;
        int vertexOffset = (int)lightVertices.size();
        lightVertices.insert(lightVertices.end(), mesh->vertices.begin(), mesh->vertices.end());
        if (mesh->normals.size() > 0)
            lightNormals.insert(lightNormals.end(), mesh->normals.begin(), mesh->normals.end());
        else
        {
            for (int j = 0; j < mesh->vertices.size(); j++)
                lightNormals.push_back(make_float3(0.0f));
        }

        for (int j = 0; j < mesh->indices.size(); j++)
        {
            uint3 index = mesh->indices[j];
            lightIndices.push_back(index + vertexOffset);
            lightEmissions.push_back(mesh->mat.Emission());
        }
    }
    int numTriangles = (int)lightIndices.size();
    std::vector<float> lightAccumArea(numTriangles);
    float totalArea = 0.0f;
    for (int i = 0; i < numTriangles; i++)
    {
        float3 v0 = lightVertices[lightIndices[i].x];
        float3 v1 = lightVertices[lightIndices[i].y];
        float3 v2 = lightVertices[lightIndices[i].z];
        float3 e0 = v1 - v0;
        float3 e1 = v2 - v0;
        float3 normal = cross(e0, e1);
        totalArea += length(normal) * 0.5f;
        lightAccumArea[i] = totalArea;
    }

    lightVertexBuffer.resize_and_copy_from_host(lightVertices);
    lightNormalBuffer.resize_and_copy_from_host(lightNormals);
    lightIndexBuffer.resize_and_copy_from_host(lightIndices);
    lightAccumAreaBuffer.resize_and_copy_from_host(lightAccumArea);
    lightEmissionBuffer.resize_and_copy_from_host(lightEmissions);
    light.Set(numTriangles, lightVertexBuffer.data(), lightNormalBuffer.data(), lightIndexBuffer.data(),
        lightAccumAreaBuffer.data(), lightEmissionBuffer.data(), totalArea);
}

OptixRayTracer::OptixRayTracer(const std::vector<std::string>& _ptxfiles, const Model* _model): model(_model)
{
    std::filesystem::path ptxFolder("ptx");
    for (const auto& ptxfile : _ptxfiles)
    {
        std::filesystem::path ptxPath = ptxFolder / ptxfile;
        std::string shader = getPtxFromFile(ptxPath.string());
        modulePTXs.push_back(shader);
    }

    std::cout << "initializing optix ..." << std::endl;
    initOptix();

    std::cout << "creating optix context ..." << std::endl;
    createContext();

    std::cout << "setting up optix pipline ..." << std::endl;
    createPipelines();

    std::cout << "building acceleration structure ..." << std::endl;
    buildAccel();

    std::cout << "creating textures ..." << std::endl;
    createTextures();

    std::cout << "building SBT ..." << std::endl;
    buildSBT();

    std::cout << "generating light ..." << std::endl;
    generateLight();

    std::cout << "Optix 7 Renderer fully set up!" << std::endl;
}