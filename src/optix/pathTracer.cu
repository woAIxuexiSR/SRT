#include "hip/hip_runtime.h"
#include "pathTracer.h"

PathTracer::PathTracer(const Model* _model, int _w, int _h) 
    : OptixRayTracer({"pathTracer.ptx"}, _model, _w, _h), launchParams(_w, _h, traversable)
{
    launchParams.SPP = 500;
    launchParams.MAX_DEPTH = 20;
    launchParams.background = make_float3(0.0f, 0.0f, 0.0f);
    launchParams.light = light;
}

void PathTracer::render(std::shared_ptr<Camera> camera, std::shared_ptr<Film> film)
{
    launchParams.colorBuffer = film->getfPtr();
    launchParams.camera = *camera;

    GPUMemory<LaunchParams<int> > launchParamsBuffer;
    launchParamsBuffer.resize_and_copy_from_host(&launchParams, 1);

    TICK(render);
    OPTIX_CHECK(optixLaunch(
        pipelines[0],
        stream,
        (hipDeviceptr_t)launchParamsBuffer.data(),
        launchParamsBuffer.size() * sizeof(LaunchParams<int>),
        &sbts[0],
        width,
        height,
        1));
    checkCudaErrors(hipDeviceSynchronize());
    TOCK(render);

    launchParams.frameId++;
}