#include "hip/hip_runtime.h"
#include "optix_ray_tracer.h"
#include <optix_function_table_definition.h>

string get_ptx_from_file(const string& filename)
{
    std::ifstream fs(filename, std::ios::in);
    if (!fs.is_open())
    {
        cout << "ERROR::Failed to open file: " << filename << endl;
        exit(-1);
    }

    std::stringstream ss;
    ss << fs.rdbuf();
    fs.close();

    return ss.str();
}

void OptixRayTracer::init_optix()
{
    checkCudaErrors(hipFree(0));

    int num_devices;
    checkCudaErrors(hipGetDeviceCount(&num_devices));
    if (num_devices == 0)
    {
        cout << "ERROR::No CUDA capable devices found!" << endl;
        return;
    }
    cout << "Found " << num_devices << " CUDA devices" << endl;

    OPTIX_CHECK(optixInit());
    cout << "Successfully initialized optix" << endl;
}

void OptixRayTracer::create_context()
{
    const int device_id = 0;
    checkCudaErrors(hipSetDevice(device_id));
    checkCudaErrors(hipStreamCreate(&stream));

    hipDeviceProp_t device_prop;
    checkCudaErrors(hipGetDeviceProperties(&device_prop, device_id));
    cout << "Running on device: " << device_prop.name << endl;

    hipCtx_t cuda_context;
    hipError_t cu_res = hipCtxGetCurrent(&cuda_context);
    if (cu_res != hipSuccess)
    {
        cout << "ERROR::Failed to get current CUDA context" << endl;
        return;
    }

    OPTIX_CHECK(optixDeviceContextCreate(cuda_context, 0, &context));
}

void OptixRayTracer::create_module(const string& ptx)
{
    OptixModule module;

    OptixModuleCompileOptions module_compile_options = {};
    module_compile_options.maxRegisterCount = OPTIX_COMPILE_DEFAULT_MAX_REGISTER_COUNT;
    module_compile_options.optLevel = OPTIX_COMPILE_OPTIMIZATION_DEFAULT;
    module_compile_options.debugLevel = OPTIX_COMPILE_DEBUG_LEVEL_DEFAULT;

    // create module
    char log[2048];
    size_t log_size = sizeof(log);
    OPTIX_CHECK(optixModuleCreateFromPTX(
        context,
        &module_compile_options,
        &pipeline_compile_options,
        ptx.c_str(),
        ptx.size(),
        log,
        &log_size,
        &module
    ));
    if (log_size > 1)
        cout << "Optix module log: " << log << endl;

    ModuleProgramGroup module_pg;

    // create raygen program
    {
        OptixProgramGroupOptions pg_options = {};
        OptixProgramGroupDesc pg_desc = {};

        pg_desc.kind = OPTIX_PROGRAM_GROUP_KIND_RAYGEN;
        pg_desc.raygen.module = module;
        pg_desc.raygen.entryFunctionName = "__raygen__";

        char log[2048];
        size_t log_size = sizeof(log);
        OPTIX_CHECK(optixProgramGroupCreate(
            context,
            &pg_desc,
            1,
            &pg_options,
            log,
            &log_size,
            &module_pg.raygenPG
        ));
        if (log_size > 1)
            cout << "Optix raygen program log: " << log << endl;
    }

    // create miss program
    {
        string name[2] = { "__miss__radiance", "__miss__shadow" };
        for (int i = 0; i < 2; i++)
        {
            OptixProgramGroupOptions pg_options = {};
            OptixProgramGroupDesc pg_desc = {};

            pg_desc.kind = OPTIX_PROGRAM_GROUP_KIND_MISS;
            pg_desc.miss.module = module;
            pg_desc.miss.entryFunctionName = name[i].c_str();

            char log[2048];
            size_t log_size = sizeof(log);
            OPTIX_CHECK(optixProgramGroupCreate(
                context,
                &pg_desc,
                1,
                &pg_options,
                log,
                &log_size,
                &module_pg.missPGs[i]
            ));
            if (log_size > 1)
                cout << "Optix miss program log: " << log << endl;
        }
    }

    // create hitgroup program
    {
        string name[2] = { "__closesthit__radiance", "__closesthit__shadow" };
        for (int i = 0; i < 2; i++)
        {
            OptixProgramGroupOptions pg_options = {};
            OptixProgramGroupDesc pg_desc = {};

            pg_desc.kind = OPTIX_PROGRAM_GROUP_KIND_HITGROUP;
            pg_desc.hitgroup.moduleCH = module;
            pg_desc.hitgroup.entryFunctionNameCH = name[i].c_str();
            pg_desc.hitgroup.moduleAH = nullptr;
            pg_desc.hitgroup.entryFunctionNameAH = nullptr;

            char log[2048];
            size_t log_size = sizeof(log);
            OPTIX_CHECK(optixProgramGroupCreate(
                context,
                &pg_desc,
                1,
                &pg_options,
                log,
                &log_size,
                &module_pg.hitgroupPGs[i]
            ));
            if (log_size > 1)
                cout << "Optix hitgroup program log: " << log << endl;
        }
    }

    module_pgs.push_back(module_pg);
}

void OptixRayTracer::create_pipeline(const vector<string>& ptxs)
{
    pipeline_compile_options = {};
    pipeline_compile_options.traversableGraphFlags = OPTIX_TRAVERSABLE_GRAPH_FLAG_ALLOW_SINGLE_GAS;
    pipeline_compile_options.usesMotionBlur = false;
    pipeline_compile_options.numPayloadValues = 2;
    pipeline_compile_options.numAttributeValues = 2;
    pipeline_compile_options.exceptionFlags = OPTIX_EXCEPTION_FLAG_NONE;
    pipeline_compile_options.pipelineLaunchParamsVariableName = "params";

    OptixPipelineLinkOptions pipeline_link_options = {};
    pipeline_link_options.maxTraceDepth = 2;

    pipelines.resize(ptxs.size());
    for (size_t i = 0; i < ptxs.size();i++)
    {
        create_module(ptxs[i]);

        char log[2048];
        size_t log_size = sizeof(log);
        OPTIX_CHECK(optixPipelineCreate(
            context,
            &pipeline_compile_options,
            &pipeline_link_options,
            (OptixProgramGroup*)&module_pgs[i],
            5,
            log,
            &log_size,
            &pipelines[i]
        ));
        if (log_size > 1)
            cout << "Optix pipeline log: " << log << endl;

        OPTIX_CHECK(optixPipelineSetStackSize(pipelines[i], 2048, 2048, 2048, 1));
    }
}

void OptixRayTracer::build_as()
{
    int mesh_num = (int)scene->meshes.size();
    DeviceSceneData& d_scene = scene->d_scene;

    vector<OptixBuildInput> triangle_input(mesh_num);
    vector<hipDeviceptr_t> d_vertices(mesh_num);
    vector<hipDeviceptr_t> d_indices(mesh_num);
    vector<uint32_t> triangle_input_flags(mesh_num);

    for (int i = 0; i < mesh_num; i++)
    {
        triangle_input[i] = {};
        triangle_input[i].type = OPTIX_BUILD_INPUT_TYPE_TRIANGLES;

        d_vertices[i] = (hipDeviceptr_t)d_scene.vertex_buffer[i].data();
        d_indices[i] = (hipDeviceptr_t)d_scene.index_buffer[i].data();

        triangle_input[i].triangleArray.vertexFormat = OPTIX_VERTEX_FORMAT_FLOAT3;
        triangle_input[i].triangleArray.vertexStrideInBytes = sizeof(float3);
        triangle_input[i].triangleArray.numVertices = (unsigned)scene->meshes[i]->vertices.size();
        triangle_input[i].triangleArray.vertexBuffers = &d_vertices[i];

        triangle_input[i].triangleArray.indexFormat = OPTIX_INDICES_FORMAT_UNSIGNED_INT3;
        triangle_input[i].triangleArray.indexStrideInBytes = sizeof(int3);
        triangle_input[i].triangleArray.numIndexTriplets = (unsigned)scene->meshes[i]->indices.size();
        triangle_input[i].triangleArray.indexBuffer = d_indices[i];

        triangle_input_flags[i] = OPTIX_GEOMETRY_FLAG_DISABLE_ANYHIT;
        triangle_input[i].triangleArray.flags = &triangle_input_flags[i];
        triangle_input[i].triangleArray.numSbtRecords = 1;
        triangle_input[i].triangleArray.sbtIndexOffsetBuffer = 0;
        triangle_input[i].triangleArray.sbtIndexOffsetSizeInBytes = 0;
        triangle_input[i].triangleArray.sbtIndexOffsetStrideInBytes = 0;
    }

    OptixAccelBuildOptions accel_options = {};
    accel_options.buildFlags = OPTIX_BUILD_FLAG_ALLOW_COMPACTION;
    accel_options.motionOptions.numKeys = 1;  // disable motion
    accel_options.operation = OPTIX_BUILD_OPERATION_BUILD;

    OptixAccelBufferSizes blas_buffer_sizes;
    OPTIX_CHECK(optixAccelComputeMemoryUsage(
        context,
        &accel_options,
        triangle_input.data(),
        (unsigned)mesh_num,
        &blas_buffer_sizes
    ));

    GPUMemory<uint64_t> compacted_size_buffer(1);
    OptixAccelEmitDesc emit_desc;
    emit_desc.type = OPTIX_PROPERTY_TYPE_COMPACTED_SIZE;
    emit_desc.result = (hipDeviceptr_t)compacted_size_buffer.data();

    GPUMemory<unsigned char> temp_buffer(blas_buffer_sizes.tempSizeInBytes);
    GPUMemory<unsigned char> output_buffer(blas_buffer_sizes.outputSizeInBytes);

    OPTIX_CHECK(optixAccelBuild(
        context,
        stream,
        &accel_options,
        triangle_input.data(),
        (unsigned)mesh_num,
        (hipDeviceptr_t)temp_buffer.data(),
        blas_buffer_sizes.tempSizeInBytes,
        (hipDeviceptr_t)output_buffer.data(),
        blas_buffer_sizes.outputSizeInBytes,
        &traversable,
        &emit_desc,
        1
    ));
    checkCudaErrors(hipDeviceSynchronize());

    uint64_t compacted_size;
    compacted_size_buffer.copy_to_host(&compacted_size);

    as_buffer.resize(compacted_size);
    OPTIX_CHECK(optixAccelCompact(
        context,
        stream,
        traversable,
        (hipDeviceptr_t)as_buffer.data(),
        compacted_size,
        &traversable
    ));
    checkCudaErrors(hipDeviceSynchronize());
}

void OptixRayTracer::build_sbt()
{
    sbts.resize(module_pgs.size());
    raygen_sbt.resize(module_pgs.size());
    miss_sbt.resize(module_pgs.size());
    hitgroup_sbt.resize(module_pgs.size());

    DeviceSceneData& d_scene = scene->d_scene;

    for (int i = 0; i < module_pgs.size(); i++)
    {
        RaygenSBTRecord raygen_record;
        OPTIX_CHECK(optixSbtRecordPackHeader(module_pgs[i].raygenPG, &raygen_record));
        raygen_sbt[i].resize_and_copy_from_host(&raygen_record, 1);
        sbts[i].raygenRecord = (hipDeviceptr_t)raygen_sbt[i].data();

        vector<MissSBTRecord> miss_records;
        for (int j = 0; j < 2; j++)
        {
            MissSBTRecord miss_record;
            OPTIX_CHECK(optixSbtRecordPackHeader(module_pgs[i].missPGs[j], &miss_record));
            miss_records.push_back(miss_record);
        }
        miss_sbt[i].resize_and_copy_from_host(miss_records);
        sbts[i].missRecordBase = (hipDeviceptr_t)miss_sbt[i].data();
        sbts[i].missRecordStrideInBytes = sizeof(MissSBTRecord);
        sbts[i].missRecordCount = (unsigned)miss_records.size();

        int mesh_num = (int)scene->meshes.size();
        vector<HitgroupSBTRecord> hitgroup_records;
        for (int k = 0; k < mesh_num; k++)
        {
            for (int j = 0; j < 2; j++)
            {
                HitgroupSBTRecord hitgroup_record;
                OPTIX_CHECK(optixSbtRecordPackHeader(module_pgs[i].hitgroupPGs[j], &hitgroup_record));

                hitgroup_record.data.vertex = (float3*)d_scene.vertex_buffer[k].data();
                hitgroup_record.data.index = (uint3*)d_scene.index_buffer[k].data();
                hitgroup_record.data.normal = (float3*)d_scene.normal_buffer[k].data();
                hitgroup_record.data.texcoord = (float2*)d_scene.texcoord_buffer[k].data();
                hitgroup_record.data.mesh_id = k;
                hitgroup_record.data.light_id = d_scene.meshid_to_lightid[k];
                hitgroup_record.data.mat = d_scene.material_buffer.data() + scene->meshes[k]->material_id;

                if (scene->meshes[k]->texture_id >= 0)
                {
                    hitgroup_record.data.has_texture = true;
                    hitgroup_record.data.texture = d_scene.texture_objects[scene->meshes[k]->texture_id];
                }
                else
                    hitgroup_record.data.has_texture = false;

                hitgroup_records.push_back(hitgroup_record);
            }
        }
        hitgroup_sbt[i].resize_and_copy_from_host(hitgroup_records);
        sbts[i].hitgroupRecordBase = (hipDeviceptr_t)hitgroup_sbt[i].data();
        sbts[i].hitgroupRecordStrideInBytes = sizeof(HitgroupSBTRecord);
        sbts[i].hitgroupRecordCount = (unsigned)hitgroup_records.size();
    }
}

OptixRayTracer::OptixRayTracer(const vector<string>& _ptxfiles, shared_ptr<Scene> _scene)
    : scene(_scene)
{
    cout << "Initializing optix..." << endl;
    init_optix();

    cout << "Creating optix context..." << endl;
    create_context();

    cout << "Creating optix pipeline..." << endl;
    std::filesystem::path ptx_folder("ptx");
    vector<string> ptxs;
    for (const auto& ptxfile : _ptxfiles)
    {
        std::filesystem::path ptx_path = ptx_folder / ptxfile;
        string shader = get_ptx_from_file(ptx_path.string());
        ptxs.push_back(shader);
    }
    create_pipeline(ptxs);

    cout << "Building acceleration structure..." << endl;
    build_as();

    cout << "Building shader binding table..." << endl;
    build_sbt();

    cout << "Optix fully set up!" << endl;
}